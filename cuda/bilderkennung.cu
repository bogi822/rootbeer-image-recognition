#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define NUMBER_OF_THREADS 100
#define NUMBER_OF_QUERY_IPS 100
#define NUMBER_OF_DATABASE_IPS 1000000

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

__global__ void bruteForceMatcher(float *x1, float *y1, float *strength1, float *trace1, bool *sign1, float *scale1, float *descriptors1,
		float *x2, float *y2, float *strength2, float *trace2, bool *sign2, float *scale2, float *descriptors2, int *anzahlMatches) {

    float distance, delta;

    int blockSizeQuery = NUMBER_OF_QUERY_IPS / NUMBER_OF_THREADS;
    int blockSizeDB = NUMBER_OF_DATABASE_IPS / NUMBER_OF_THREADS;

    int startIndexQuery = blockIdx.x * blockSizeQuery;
    int endIndexQuery = blockIdx.x * blockSizeQuery + blockSizeQuery;

    int startIndexDB = blockIdx.x * blockSizeDB;
    int endIndexDB =  blockIdx.x * blockSizeDB + blockSizeDB;

	for(int i = startIndexQuery; i < endIndexQuery; i++) {
	    float bestDistance = 999999999.0f;
	    float secondBestDistance = 999999999.0f;
	    for (int a = startIndexDB; a < endIndexDB; a++) {
	        if (sign1[i] == sign2[a]){
	            distance = 0;
	            for (int b = 0; b < 64; b++) {
	                delta = descriptors1[b + (i * 64)] - descriptors2[b + (a * 64)];
	                distance += (delta * delta);
	            }
	            if(distance < bestDistance){
	                secondBestDistance = bestDistance;
	                bestDistance = distance;
	            }
	        }
	    }

	    // Ermittle die beste und die 2. beste distanz
	    if (bestDistance < 0.6f * secondBestDistance) {
	    	anzahlMatches[blockIdx.x]++;
	    }
	}
}

//__global__ void add(int *a, int *b, int *c, int **d) {
//	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
//}

void random_ints(int* a, int anzahl){
   int i;
   for (i = 0; i < anzahl; ++i){
	  //a[i] = rand();
	  a[i] = 0;
   }
}

void random_floats(float* a, int anzahl){
   int i;
   for (i = 0; i < anzahl; ++i){
	  //a[i] = rand();
	  a[i] = 0.0005f;
   }
}

void random_bools(bool* a, int anzahl){
   int i;
   for (i = 0; i < anzahl; ++i){
	  a[i] = true;
   }
}

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(void) {

	printf("Start ...\n");

	// Host variables
	float *x1, *y1, *strength1, *trace1, *scale1, *descriptors1, *x2, *y2, *strength2, *trace2, *scale2, *descriptors2;
	bool *sign1, *sign2;
	int *anzahlMatches;

	// Device variables
	float *d_x1, *d_y1, *d_strength1, *d_trace1, *d_scale1, *d_descriptors1, *d_x2, *d_y2, *d_strength2, *d_trace2, *d_scale2, *d_descriptors2;
	bool *d_sign1, *d_sign2;
	int *d_anzahlMatches;

	// Allocate cuda space
	printf("Start allocating space for cuda...\n");
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_x1, NUMBER_OF_QUERY_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_y1, NUMBER_OF_QUERY_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_strength1, NUMBER_OF_QUERY_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_trace1, NUMBER_OF_QUERY_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_scale1, NUMBER_OF_QUERY_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_descriptors1, NUMBER_OF_QUERY_IPS * sizeof(float) * 64));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_sign1, NUMBER_OF_QUERY_IPS * sizeof(bool)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_x2, NUMBER_OF_DATABASE_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_y2, NUMBER_OF_DATABASE_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_strength2, NUMBER_OF_DATABASE_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_trace2, NUMBER_OF_DATABASE_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_scale2, NUMBER_OF_DATABASE_IPS * sizeof(float)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_descriptors2, NUMBER_OF_DATABASE_IPS * sizeof(float) * 64));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_sign2, NUMBER_OF_DATABASE_IPS * sizeof(bool)));
	CUDA_CHECK_RETURN(hipMalloc((void** ) &d_anzahlMatches, NUMBER_OF_THREADS * sizeof(int)));
	printf("Allocating space for cuda finished!\n");

	// Create space for the query-data
	printf("Start allocating space for host...\n");
	x1 = (float *)malloc(NUMBER_OF_QUERY_IPS * sizeof(float)); random_floats(x1, NUMBER_OF_QUERY_IPS);
	y1 = (float *)malloc(NUMBER_OF_QUERY_IPS * sizeof(float)); random_floats(y1, NUMBER_OF_QUERY_IPS);
	strength1 = (float *)malloc(NUMBER_OF_QUERY_IPS * sizeof(float)); random_floats(strength1, NUMBER_OF_QUERY_IPS);
	trace1 = (float *)malloc(NUMBER_OF_QUERY_IPS * sizeof(float)); random_floats(trace1, NUMBER_OF_QUERY_IPS);
	scale1 = (float *)malloc(NUMBER_OF_QUERY_IPS * sizeof(float)); random_floats(scale1, NUMBER_OF_QUERY_IPS);
	descriptors1 = (float *)malloc(NUMBER_OF_QUERY_IPS * sizeof(float) * 64); random_floats(descriptors1, NUMBER_OF_QUERY_IPS * 64);
	sign1 = (bool *)malloc(NUMBER_OF_QUERY_IPS * sizeof(bool)); random_bools(sign1, NUMBER_OF_QUERY_IPS);

	x2 = (float *)malloc(NUMBER_OF_DATABASE_IPS * sizeof(float)); random_floats(x2, NUMBER_OF_DATABASE_IPS);
	y2 = (float *)malloc(NUMBER_OF_DATABASE_IPS * sizeof(float)); random_floats(y2, NUMBER_OF_DATABASE_IPS);
	strength2 = (float *)malloc(NUMBER_OF_DATABASE_IPS * sizeof(float)); random_floats(strength2, NUMBER_OF_DATABASE_IPS);
	trace2 = (float *)malloc(NUMBER_OF_DATABASE_IPS * sizeof(float)); random_floats(trace2, NUMBER_OF_DATABASE_IPS);
	scale2 = (float *)malloc(NUMBER_OF_DATABASE_IPS * sizeof(float)); random_floats(scale2, NUMBER_OF_DATABASE_IPS);
	descriptors2 = (float *)malloc(NUMBER_OF_DATABASE_IPS * sizeof(float) * 64); random_floats(descriptors2, NUMBER_OF_DATABASE_IPS * 64);
	sign2 = (bool *)malloc(NUMBER_OF_DATABASE_IPS * sizeof(bool)); random_bools(sign2, NUMBER_OF_DATABASE_IPS);
	anzahlMatches = (int *)malloc(NUMBER_OF_THREADS * sizeof(int));

	// Zero-fill the current match count
	for(int i = 0; i < NUMBER_OF_THREADS; i++){
		anzahlMatches[i] = 0;
	}

	printf("Start allocating space for host finished!\n");

	// Copy data to device
	printf("Start copying stuff to the gpu...\n");
	CUDA_CHECK_RETURN(hipMemcpy(d_x1, x1, NUMBER_OF_QUERY_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_y1, y1, NUMBER_OF_QUERY_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_strength1, strength1, NUMBER_OF_QUERY_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_trace1, trace1, NUMBER_OF_QUERY_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_scale1, scale1, NUMBER_OF_QUERY_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_descriptors1, descriptors1, NUMBER_OF_QUERY_IPS * sizeof(float) * 64, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_sign1, sign1, NUMBER_OF_QUERY_IPS * sizeof(bool), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_x2, x2, NUMBER_OF_DATABASE_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_y2, y2, NUMBER_OF_DATABASE_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_strength2, strength2, NUMBER_OF_DATABASE_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_trace2, trace2, NUMBER_OF_DATABASE_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_scale2, scale2, NUMBER_OF_DATABASE_IPS * sizeof(float), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_descriptors2, descriptors2, NUMBER_OF_DATABASE_IPS * sizeof(float) * 64, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_sign2, sign2, NUMBER_OF_DATABASE_IPS * sizeof(bool), hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_anzahlMatches, anzahlMatches, NUMBER_OF_THREADS * sizeof(int), hipMemcpyHostToDevice));
	printf("Copying stuff to the gpu finished!\n");

	printf("Start image-recognition ...\n");

	bruteForceMatcher<<<NUMBER_OF_THREADS, 1>>>(d_x1, d_y1, d_strength1, d_trace1, d_sign1, d_scale1, d_descriptors1,
								d_x2, d_y2, d_strength2, d_trace2, d_sign2, d_scale2, d_descriptors2, d_anzahlMatches);

	printf("Image-recognition ready!");

	CUDA_CHECK_RETURN(hipDeviceSynchronize());	// Wait for the GPU launched work to complete
	CUDA_CHECK_RETURN(hipGetLastError());

	CUDA_CHECK_RETURN(hipMemcpy(anzahlMatches, d_anzahlMatches, NUMBER_OF_THREADS * sizeof(int), hipMemcpyDeviceToHost));

	for(int i = 0; i < NUMBER_OF_THREADS; i++){
		printf("Ergebnis: %d\n", anzahlMatches[i]);
	}

	CUDA_CHECK_RETURN(hipFree(d_x1));
	CUDA_CHECK_RETURN(hipFree(d_y1));
	CUDA_CHECK_RETURN(hipFree(d_strength1));
	CUDA_CHECK_RETURN(hipFree(d_trace1));
	CUDA_CHECK_RETURN(hipFree(d_scale1));
	CUDA_CHECK_RETURN(hipFree(d_descriptors1));
	CUDA_CHECK_RETURN(hipFree(d_sign1));
	CUDA_CHECK_RETURN(hipFree(d_x2));
	CUDA_CHECK_RETURN(hipFree(d_y2));
	CUDA_CHECK_RETURN(hipFree(d_strength2));
	CUDA_CHECK_RETURN(hipFree(d_trace2));
	CUDA_CHECK_RETURN(hipFree(d_scale2));
	CUDA_CHECK_RETURN(hipFree(d_descriptors2));
	CUDA_CHECK_RETURN(hipFree(d_sign2));

	CUDA_CHECK_RETURN(hipDeviceReset());
	printf("End!\n");
	return 0;
}


